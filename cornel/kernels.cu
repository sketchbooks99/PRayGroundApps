#include "hip/hip_runtime.h"
#pragma once 

#include <prayground/optix/cuda/device_util.cuh>

#include <prayground/core/bsdf.h>
#include <prayground/core/onb.h>
#include <prayground/core/color.h>
#include <prayground/core/interaction.h>
#include <prayground/core/ray.h>

#include <prayground/shape/trianglemesh.h>
#include <prayground/shape/plane.h>
#include <prayground/shape/sphere.h>

#include <prayground/texture/constant.h>

#include <prayground/material/diffuse.h>
#include <prayground/material/dielectric.h>

#include <prayground/emitter/area.h>
#include <prayground/emitter/envmap.h>

#include "params.h"

using namespace prayground;

extern "C" {
__constant__ LaunchParams params;
}

INLINE DEVICE SurfaceInteraction* getSurfaceInteraction()
{
    const uint32_t u0 = getPayload<0>();
    const uint32_t u1 = getPayload<1>();
    return reinterpret_cast<SurfaceInteraction*>(unpackPointer(u0, u1));
}

INLINE DEVICE void trace(
    OptixTraversableHandle handle,
    float3                 ray_origin,
    float3                 ray_direction,
    float                  tmin,
    float                  tmax,
    unsigned int           ray_type,
    SurfaceInteraction*    si
) 
{
    unsigned int u0, u1;
    packPointer( si, u0, u1 );
    optixTrace(
        handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,                // rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        ray_type,        
        1,           
        ray_type,        
        u0, u1 );	
}

static __forceinline__ __device__ void getCameraRay(const CameraData& camera, const float x, const float y, float3& ro, float3& rd)
{
    rd = normalize(x * camera.U + y * camera.V + camera.W);
    ro = camera.origin;
}

extern "C" __device__ void __raygen__pinhole()
{
    const RaygenData* raygen = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    const int frame = params.frame;
    const uint3 idx = optixGetLaunchIndex();
    unsigned int seed = tea<4>(idx.x * params.width + idx.y, frame);

    float3 result = make_float3(0.0f);

    int i = params.samples_per_launch;

    do
    {
        const float2 subpixel_jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

        const float2 d = 2.0f * make_float2(
            (static_cast<float>(idx.x) + subpixel_jitter.x) / static_cast<float>(params.width),
            (static_cast<float>(idx.y) + subpixel_jitter.y) / static_cast<float>(params.height)
        ) - 1.0f;

        float3 ro, rd;
        getCameraRay(raygen->camera, d.x, d.y, ro, rd);

        float3 throughput = make_float3(1.0f);

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = make_float3(0.0f);
        si.albedo = make_float3(0.0f);
        si.trace_terminate = false;
        si.radiance_evaled = false;

        int depth = 0;
        for ( ;; ) {

            if ( depth >= params.max_depth )
				break;

            trace(params.handle, ro, rd, 0.01f, 1e16f, 0, &si);

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }

            optixDirectCall<void, SurfaceInteraction*, void*>(
                si.surface_info.sample_id, 
                &si, 
                si.surface_info.data
            );

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }
            else 
            {
                throughput *= si.albedo;
            }
            
            ro = si.p;
            rd = si.wo;

            ++depth;
        }
    } while (--i);

    const uint3 launch_index = optixGetLaunchIndex();
    const unsigned int image_index = launch_index.y * params.width + launch_index.x;

    if (result.x != result.x) result.x = 0.0f;
    if (result.y != result.y) result.y = 0.0f;
    if (result.z != result.z) result.z = 0.0f;

    float3 accum_color = result / static_cast<float>(params.samples_per_launch);

    if (frame > 0)
    {
        const float a = 1.0f / static_cast<float>(frame + 1);
        const float3 accum_color_prev = make_float3(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_index] = make_float4(accum_color, 1.0f);
    uchar3 color = make_color(accum_color);
    params.result_buffer[image_index] = make_uchar4(color.x, color.y, color.z, 255);
}

extern "C" __device__ void __miss__envmap()
{
    MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    EnvironmentEmitterData* env = reinterpret_cast<EnvironmentEmitterData*>(data->env_data);
    SurfaceInteraction* si = getSurfaceInteraction();

    Ray ray = getWorldRay();

    const float a = dot(ray.d, ray.d);
    const float half_b = dot(ray.o, ray.d);
    const float c = dot(ray.o, ray.o) - 1e8f*1e8f;
    const float discriminant = half_b * half_b - a*c;

    float sqrtd = sqrtf(discriminant);
    float t = (-half_b + sqrtd) / a;

    float3 p = normalize(ray.at(t));

    float phi = atan2(p.z, p.x);
    float theta = asin(p.y);
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    si->uv = make_float2(u, v);
    si->trace_terminate = true;
    si->surface_info.type = SurfaceType::None;
    si->emission = optixDirectCall<float3, const float2&, void*>(
        env->tex_program_id, si->uv, env->tex_data
        );
}

// Hitgroups
extern "C" __device__ void __closesthit__mesh()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const MeshData* mesh_data = reinterpret_cast<MeshData*>(data->shape_data);

    Ray ray = getWorldRay();
    
    const int prim_id = optixGetPrimitiveIndex();
    const Face face = mesh_data->faces[prim_id];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    const float2 texcoord0 = mesh_data->texcoords[face.texcoord_id.x];
    const float2 texcoord1 = mesh_data->texcoords[face.texcoord_id.y];
    const float2 texcoord2 = mesh_data->texcoords[face.texcoord_id.z];
    const float2 texcoords = (1-u-v)*texcoord0 + u*texcoord1 + v*texcoord2;

    float3 n0 = mesh_data->normals[face.normal_id.x];
	float3 n1 = mesh_data->normals[face.normal_id.y];
	float3 n2 = mesh_data->normals[face.normal_id.z];

    // Linear interpolation of normal by barycentric coordinates.
    float3 local_n = (1.0f-u-v)*n0 + u*n1 + v*n2;
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = texcoords;
    si->surface_info = data->surface_info;
}

extern "C" __device__ void __intersection__plane()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const PlaneData* plane_data = reinterpret_cast<PlaneData*>(data->shape_data);

    const float2 min = plane_data->min;
    const float2 max = plane_data->max;

    Ray ray = getLocalRay();

    const float t = -ray.o.y / ray.d.y;

    const float x = ray.o.x + t * ray.d.x;
    const float z = ray.o.z + t * ray.d.z;

    float2 uv = make_float2((x - min.x) / (max.x - min.x), (z - min.y) / (max.y - min.y));

    float3 n = make_float3(0, 1, 0);

    if (min.x < x && x < max.x && min.y < z && z < max.y && ray.tmin < t && t < ray.tmax)
        optixReportIntersection(t, 0, float3_as_ints(n), float2_as_ints(uv));
}

extern "C" __device__ void __closesthit__plane()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    
    Ray ray = getWorldRay();

    float3 local_n = getFloat3FromAttribute<0>();
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);
    float2 uv = getFloat2FromAttribute<3>();

    SurfaceInteraction* si = getSurfaceInteraction();

    si->p = ray.at(ray.tmax);
    si->n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = uv;
    si->surface_info = data->surface_info;
}

static __forceinline__ __device__ float2 getSphereUV(const float3& p) {
    float phi = atan2(p.z, p.x);
    if (phi < 0) phi += 2.0f * math::pi;
    float theta = acos(p.y);
    float u = phi / (2.0f * math::pi);
    float v = theta / math::pi;
    return make_float2(u, v);
}

extern "C" __device__ void __intersection__sphere() {
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere_data = reinterpret_cast<SphereData*>(data->shape_data);

    const float3 center = sphere_data->center;
    const float radius = sphere_data->radius;

    Ray ray = getLocalRay();

    const float3 oc = ray.o - center;
    const float a = dot(ray.d, ray.d);
    const float half_b = dot(oc, ray.d);
    const float c = dot(oc, oc) - radius * radius;
    const float discriminant = half_b * half_b - a * c;

    if (discriminant > 0.0f) {
        float sqrtd = sqrtf(discriminant);
        float t1 = (-half_b - sqrtd) / a;
        bool check_second = true;
        if (t1 > ray.tmin && t1 < ray.tmax) {
            float3 normal = normalize((ray.at(t1) - center) / radius);
            const float2 uv = getSphereUV(normal);
            check_second = false;
            optixReportIntersection(t1, 0, float3_as_ints(normal), float2_as_ints(uv));
        }

        if (check_second) {
            float t2 = (-half_b + sqrtd) / a;
            if (t2 > ray.tmin && t2 < ray.tmax) {
                float3 normal = normalize((ray.at(t2) - center) / radius);
                const float2 uv = getSphereUV(normal);
                optixReportIntersection(t2, 0, float3_as_ints(normal), float2_as_ints(uv));
            }
        }
    }
}

extern "C" __device__ void __closesthit__sphere() {
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere_data = reinterpret_cast<SphereData*>(data->shape_data);

    Ray ray = getWorldRay();

    float3 local_n = getFloat3FromAttribute<0>();
    float2 uv = getFloat2FromAttribute<3>();
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->n = world_n;
    si->t = ray.tmax;
    si->wi = ray.d;
    si->uv = uv;
    si->surface_info = data->surface_info;
}

// Textures
extern "C" __device__ float3 __direct_callable__constant(const float2& uv, void* tex_data) {
    const ConstantTextureData* constant = reinterpret_cast<ConstantTextureData*>(tex_data);
    return make_float3(constant->color);
}

// Materials
extern "C" __device__ void __direct_callable__diffuse(SurfaceInteraction* si, void* mat_data)
{
    const DiffuseData* diffuse = reinterpret_cast<DiffuseData*>(mat_data);
    if (diffuse->twosided)
        si->n = faceforward(si->n, -si->wi, si->n);
    
    si->trace_terminate = false;
    uint32_t seed = si->seed;
    float3 wi = randomSampleHemisphere(seed);
    Onb onb(si->n);
    onb.inverseTransform(wi);
    si->wo = wi;
    si->seed = seed;
    si->albedo = optixDirectCall<float3, const float2&, void*>(diffuse->tex_program_id, si->uv, diffuse->tex_data);
}

extern "C" __device__ void __direct_callable__glass(SurfaceInteraction* si, void* mat_data)
{
    const DielectricData* dielectric = reinterpret_cast<DielectricData*>(mat_data);

    float ni = 1.0f; // air
    float nt = dielectric->ior;  // ior specified 
    float cosine = dot(si->wi, si->n);
    bool into = cosine < 0;
    float3 outward_normal = into ? si->n : -si->n;

    if (!into) swap(ni, nt);

    cosine = fabs(cosine);
    float sine = sqrtf(1.0 - cosine*cosine);
    bool cannot_refract = (ni / nt) * sine > 1.0f;

    float reflect_prob = fresnel(cosine, ni, nt);
    unsigned int seed = si->seed;

    if (cannot_refract || reflect_prob > rnd(seed))
        si->wo = reflect(si->wi, outward_normal);
    else    
        si->wo = refract(si->wi, outward_normal, cosine, ni, nt);
    si->radiance_evaled = false;
    si->trace_terminate = false;
    si->seed = seed;
    si->albedo = optixDirectCall<float3, const float2&, void*>(dielectric->tex_program_id, si->uv, dielectric->tex_data);
}

extern "C" __device__ void __direct_callable__area(SurfaceInteraction* si, void* mat_data)
{
    const AreaEmitterData* area = reinterpret_cast<AreaEmitterData*>(mat_data);
    si->trace_terminate = true;
    float is_emitted = dot(si->wi, si->n) < 0.0f ? 1.0f : 0.0f;
    if (area->twosided)
    {
        is_emitted = 1.0f;
        si->n = faceforward(si->n, -si->wi, si->n);
    }

    const float4 base = optixDirectCall<float4, const float2&, void*>(
        area->tex_program_id, si->uv, area->tex_data);
    si->albedo = make_float3(base);
    
    si->emission = si->albedo * area->intensity * is_emitted;
}

